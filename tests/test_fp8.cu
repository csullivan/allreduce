#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>

#include <cassert>
#include <cstring>
#include <iostream>
#include <vector>

const char* RED = "\033[31m";
const char* GREEN = "\033[32m";
const char* YELLOW = "\033[33m";
const char* BOLD_RED = "\033[1;31m";
const char* RESET = "\033[0m";

__global__ void vectorAddPTX(const float* A, const float* B, float* C, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < numElements) {
    // Inline PTX to perform the addition
    // %0, %1, and %2 correspond to C[i], A[i], and B[i] respectively
    asm("{\n\t"
        "add.f32 %0, %1, %2;\n\t"
        "}"
        : "=f"(C[i])
        : "f"(A[i]), "f"(B[i]));
  }
}

int test_ptx() {
  int numElements = 50000;
  size_t size = numElements * sizeof(float);
  float* h_A = new float[numElements];
  float* h_B = new float[numElements];
  float* h_C = new float[numElements];

  // Initialize input vectors
  for (int i = 0; i < numElements; ++i) {
    h_A[i] = static_cast<float>(i);
    h_B[i] = static_cast<float>(i * 2);
  }

  float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  vectorAddPTX<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << RED << "Failed to launch vectorAddPTX kernel (error code "
              << hipGetErrorString(err) << ")" << RESET << std::endl;
    return -1;
  }

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < numElements; ++i) {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
      std::cerr << BOLD_RED << "[FAILED]" << YELLOW << " Result verification failed at element "
                << i << RESET << std::endl;
      return -1;
    }
  }

  std::cout << GREEN << "[PASSED]" << YELLOW << " ptx_vector_add" << RESET << std::endl;

  // Free device global memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Free host memory
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;

  return 0;
}

__global__ void testFp8Conversion() {
  __half2_raw input = __floats2half2_rn(1.0f, 0.5f);
  __hip_fp8x2_storage_t fp8x2 = __hip_cvt_halfraw2_to_fp8x2(input, __HIP_SATFINITE, __HIP_E5M2_FNUZ);
  __half2_raw output = __hip_cvt_fp8x2_to_halfraw2(fp8x2, __HIP_E5M2_FNUZ);
  assert(input.x == output.x && input.y == output.y);
}

int test_fp8_conversion() {
  testFp8Conversion<<<1, 1>>>();

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << BOLD_RED << "CUDA error: " << hipGetErrorString(err) << std::endl;
    std::cerr << RED << "[FAILED]" << YELLOW << " test_fp8_conversion" << RESET << std::endl;
    return -1;
  }

  hipDeviceSynchronize();

  std::cout << GREEN << "[PASSED]" << YELLOW << " test_fp8_conversion" << RESET << std::endl;
  return 0;
}

int main(int argc, char* argv[]) {
  // test_ptx();
  test_fp8_conversion();
  return 0;
}
